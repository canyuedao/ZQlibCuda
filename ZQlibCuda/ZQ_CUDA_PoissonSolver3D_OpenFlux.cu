#include "hip/hip_runtime.h"
#ifndef _ZQ_CUDA_POISSON_SOLVER_3D_OPEN_FLUX_CU_
#define _ZQ_CUDA_POISSON_SOLVER_3D_OPEN_FLUX_CU_

#include "ZQ_CUDA_PoissonSolver3D_OpenFlux.cuh"
#include "ZQ_CUDA_ImageProcessing3D.cuh"

namespace ZQ_CUDA_PoissonSolver3D
{
	__global__
	void SolveFlux_OpenFlux_u_RedBlack_Kernel(float* out_du, const float* du, const float* dv, const float* dw, const float* divergence, const float* lambda, const float aug_coeff,
										const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x > width || y >= height)
			return ;

		int rest = x%2;

		if(rest == (redkernel ? 1 : 0))
			return;
			
		for(int z = 0;z < depth;z++)
		{
			float coeff = 2.0f,sigma = 0.0f;
			
			if(x < width)
			{
				sigma -= lambda[z*height*width+y*width+x];
				coeff += aug_coeff;
				sigma += aug_coeff*(du[z*height*(width+1)+y*(width+1)+x+1]+dv[z*(height+1)*width+(y+1)*width+x]-dv[z*(height+1)*width+y*width+x]
									+dw[(z+1)*height*width+y*width+x]-dw[z*height*width+y*width+x]+divergence[z*height*width+y*width+x]);
			}
			
			if(x > 0)
			{
				sigma += lambda[z*height*width+y*width+x-1];
				coeff += aug_coeff;
				sigma -= aug_coeff*(-du[z*height*(width+1)+y*(width+1)+x-1]+dv[z*(height+1)*width+(y+1)*width+x-1]-dv[z*(height+1)*width+y*width+x-1]
									+dw[(z+1)*height*width+y*width+x-1]-dw[z*height*width+y*width+x-1]+divergence[z*height*width+y*width+x-1]);
			}
			out_du[z*height*(width+1)+y*(width+1)+x] = sigma/coeff;
		}
	}
	
	
	__global__
	void SolveFlux_OpenFlux_v_RedBlack_Kernel(float* out_dv, const float* du, const float* dv, const float* dw, const float* divergence, const float* lambda, const float aug_coeff,
										const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= width || y > height)
			return ;

		int rest = y%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		for(int z = 0;z < depth;z++)
		{
			float coeff = 2.0f,sigma = 0.0f;
			
			if(y < height)
			{
				sigma -= lambda[z*height*width+y*width+x];
				coeff += aug_coeff;
				sigma += aug_coeff*(du[z*height*(width+1)+y*(width+1)+x+1]-du[z*height*(width+1)+y*(width+1)+x]+dv[z*(height+1)*width+(y+1)*width+x]
									+dw[(z+1)*height*width+y*width+x]-dw[z*height*width+y*width+x]+divergence[z*height*width+y*width+x]);
			}
			
			if(y > 0)
			{
				sigma += lambda[z*height*width+(y-1)*width+x];
				coeff += aug_coeff;
				sigma -= aug_coeff*(du[z*height*(width+1)+(y-1)*(width+1)+x+1]-du[z*height*(width+1)+(y-1)*(width+1)+x]-dv[z*(height+1)*width+(y-1)*width+x]
									+dw[(z+1)*height*width+(y-1)*width+x]-dw[z*height*width+(y-1)*width+x]+divergence[z*height*width+(y-1)*width+x]);
			}
			out_dv[z*(height+1)*width+y*width+x] = sigma/coeff;
		}
	}
	
	__global__
	void SolveFlux_OpenFlux_w_RedBlack_Kernel(float* out_dw, const float* du, const float* dv, const float* dw, const float* divergence, const float* lambda, const float aug_coeff,
										const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= width || y >= height)
			return ;

		int start = redkernel ? 0 : 1;

		for(int z = start;z <= depth;z += 2)
		{
			float coeff = 2.0f,sigma = 0.0f;
			
			if(z < depth)
			{
				sigma -= lambda[z*height*width+y*width+x];
				coeff += aug_coeff;
				sigma += aug_coeff*(du[z*height*(width+1)+y*(width+1)+x+1]-du[z*height*(width+1)+y*(width+1)+x]
									+dv[z*(height+1)*width+(y+1)*width+x]-dv[z*(height+1)*width+y*width+x]
									+dw[(z+1)*height*width+y*width+x]+divergence[z*height*width+y*width+x]);
			}
			
			if(z > 0)
			{
				sigma += lambda[(z-1)*height*width+y*width+x];
				coeff += aug_coeff;
				sigma -= aug_coeff*(du[(z-1)*height*(width+1)+y*(width+1)+x+1]-du[(z-1)*height*(width+1)+y*(width+1)+x]
									+dv[(z-1)*(height+1)*width+(y+1)*width+x]-dv[(z-1)*(height+1)*width+y*width+x]
									-dw[(z-1)*height*width+y*width+x]+divergence[(z-1)*height*width+y*width+x]);
			}
			out_dw[z*height*width+y*width+x] = sigma/coeff;
		}
	}
	
	
	
	__global__
	void SolveFlux_OpenFlux_occupy_u_RedBlack_Kernel(float* out_du, const float* du, const float* dv, const float* dw, const bool* occupy, const float* divergence, const float* lambda, const float aug_coeff,
										const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x > width || y >= height)
			return ;

		int rest = x%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		for(int z = 0;z < depth;z++)
		{
			float coeff = 2.0f,sigma = 0.0f;
			
			if(x < width)
			{
				if(occupy[z*height*width+y*width+x])
					continue ;
				sigma -= lambda[z*height*width+y*width+x];
				coeff += aug_coeff;
				sigma += aug_coeff*(du[z*height*(width+1)+y*(width+1)+x+1]+dv[z*(height+1)*width+(y+1)*width+x]-dv[z*(height+1)*width+y*width+x]
									+dw[(z+1)*height*width+y*width+x]-dw[z*height*width+y*width+x]+divergence[z*height*width+y*width+x]);
			}
			
			if(x > 0)
			{
				if(occupy[z*height*width+y*width+x-1])
					continue ;
				sigma += lambda[z*height*width+y*width+x-1];
				coeff += aug_coeff;
				sigma -= aug_coeff*(-du[z*height*(width+1)+y*(width+1)+x-1]+dv[z*(height+1)*width+(y+1)*width+x-1]-dv[z*(height+1)*width+y*width+x-1]
									+dw[(z+1)*height*width+y*width+x-1]-dw[z*height*width+y*width+x-1]+divergence[z*height*width+y*width+x-1]);
			}
			out_du[z*height*(width+1)+y*(width+1)+x] = sigma/coeff;
		}
	}
	
	
	__global__
	void SolveFlux_OpenFlux_occupy_v_RedBlack_Kernel(float* out_dv, const float* du, const float* dv, const float* dw, const bool* occupy, const float* divergence, const float* lambda, const float aug_coeff,
										const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= width || y > height)
			return ;

		int rest = y%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		for(int z = 0;z < depth;z++)
		{
			float coeff = 2.0f,sigma = 0.0f;
			
			if(y < height)
			{
				if(occupy[z*height*width+y*width+x])
					continue ;
				sigma -= lambda[z*height*width+y*width+x];
				coeff += aug_coeff;
				sigma += aug_coeff*(du[z*height*(width+1)+y*(width+1)+x+1]-du[z*height*(width+1)+y*(width+1)+x]+dv[z*(height+1)*width+(y+1)*width+x]
									+dw[(z+1)*height*width+y*width+x]-dw[z*height*width+y*width+x]+divergence[z*height*width+y*width+x]);
			}
			
			if(y > 0)
			{
				if(occupy[z*height*width+(y-1)*width+x])
					continue ;
				sigma += lambda[z*height*width+(y-1)*width+x];
				coeff += aug_coeff;
				sigma -= aug_coeff*(du[z*height*(width+1)+(y-1)*(width+1)+x+1]-du[z*height*(width+1)+(y-1)*(width+1)+x]-dv[z*(height+1)*width+(y-1)*width+x]
									+dw[(z+1)*height*width+(y-1)*width+x]-dw[z*height*width+(y-1)*width+x]+divergence[z*height*width+(y-1)*width+x]);
			}
			out_dv[z*(height+1)*width+y*width+x] = sigma/coeff;
		}
	}
	
	__global__
	void SolveFlux_OpenFlux_occupy_w_RedBlack_Kernel(float* out_dw, const float* du, const float* dv, const float* dw, const bool* occupy, const float* divergence, const float* lambda, const float aug_coeff,
										const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= width || y >= height)
			return ;

		int start = redkernel ? 0 : 1;
		for(int z = start;z <= depth;z += 2)
		{
			float coeff = 2.0f,sigma = 0.0f;
			
			if(z < depth)
			{
				if(occupy[z*height*width+y*width+x])
					continue ;
				sigma -= lambda[z*height*width+y*width+x];
				coeff += aug_coeff;
				sigma += aug_coeff*(du[z*height*(width+1)+y*(width+1)+x+1]-du[z*height*(width+1)+y*(width+1)+x]
									+dv[z*(height+1)*width+(y+1)*width+x]-dv[z*(height+1)*width+y*width+x]
									+dw[(z+1)*height*width+y*width+x]+divergence[z*height*width+y*width+x]);
			}
			
			if(z > 0)
			{
				if(occupy[(z-1)*height*width+y*width+x])
					continue ;
				sigma += lambda[(z-1)*height*width+y*width+x];
				coeff += aug_coeff;
				sigma -= aug_coeff*(du[(z-1)*height*(width+1)+y*(width+1)+x+1]-du[(z-1)*height*(width+1)+y*(width+1)+x]
									+dv[(z-1)*(height+1)*width+(y+1)*width+x]-dv[(z-1)*(height+1)*width+y*width+x]
									-dw[(z-1)*height*width+y*width+x]+divergence[(z-1)*height*width+y*width+x]);
			}
			out_dw[z*height*width+y*width+x] = sigma/coeff;
		}
	}
	
	
	__global__
	void SolveFlux_OpenFlux_FaceRatio_u_RedBlack_Kernel(float* out_du, const float* du, const float* dv, const float* dw, const float* unoccupyU, const float* unoccupyV, const float* unoccupyW,
										const float* divergence, const float* lambda, const float aug_coeff, const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x > width || y >= height)
			return ;

		int rest = x%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		for(int z = 0;z < depth;z++)
		{
			float ratio = unoccupyU[z*height*(width+1)+y*(width+1)+x];
			float ratio2 = ratio*ratio;
			
			if(ratio == 0)
				continue ;
			
			float coeff = 2.0f*ratio,sigma = 0.0f;
			
			if(x < width)
			{
				sigma -= ratio*lambda[z*height*width+y*width+x];
				coeff += ratio2*aug_coeff;
				sigma += ratio*aug_coeff*(
							unoccupyU[z*height*(width+1)+y*(width+1)+x+1]*du[z*height*(width+1)+y*(width+1)+x+1]
						   +unoccupyV[z*(height+1)*width+(y+1)*width+x]*dv[z*(height+1)*width+(y+1)*width+x]
						   -unoccupyV[z*(height+1)*width+y*width+x]*dv[z*(height+1)*width+y*width+x]
						   +unoccupyW[(z+1)*height*width+y*width+x]*dw[(z+1)*height*width+y*width+x]
						   -unoccupyW[z*height*width+y*width+x]*dw[z*height*width+y*width+x]
						   +divergence[z*height*width+y*width+x]);
			}
			
			if(x > 0)
			{
				sigma += ratio*lambda[z*height*width+y*width+x-1];
				coeff += ratio2*aug_coeff;
				sigma -= ratio*aug_coeff*(
							-unoccupyU[z*height*(width+1)+y*(width+1)+x-1]*du[z*height*(width+1)+y*(width+1)+x-1]
							+unoccupyV[z*(height+1)*width+(y+1)*width+x-1]*dv[z*(height+1)*width+(y+1)*width+x-1]
							-unoccupyV[z*(height+1)*width+y*width+x-1]*dv[z*(height+1)*width+y*width+x-1]
							+unoccupyW[(z+1)*height*width+y*width+x-1]*dw[(z+1)*height*width+y*width+x-1]
							-unoccupyW[z*height*width+y*width+x-1]*dw[z*height*width+y*width+x-1]
							+divergence[z*height*width+y*width+x-1]);
			}
			out_du[z*height*(width+1)+y*(width+1)+x] = sigma/coeff;
		}
	}
	
	__global__
	void SolveFlux_OpenFlux_FaceRatio_v_RedBlack_Kernel(float* out_dv, const float* du, const float* dv, const float* dw, const float* unoccupyU, const float* unoccupyV, const float* unoccupyW,
										const float* divergence, const float* lambda, const float aug_coeff, const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= width || y > height)
			return ;

		int rest = y%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		for(int z = 0;z < depth;z++)
		{
			float ratio = unoccupyV[z*(height+1)*width+y*width+x];
			
			if(ratio == 0)
				continue ;
				
			float ratio2 = ratio*ratio;
			float coeff = 2.0f*ratio,sigma = 0.0f;
			
			if(y < height)
			{
				sigma -= ratio*lambda[z*height*width+y*width+x];
				coeff += ratio2*aug_coeff;
				sigma += ratio*aug_coeff*(
							unoccupyU[z*height*(width+1)+y*(width+1)+x+1]*du[z*height*(width+1)+y*(width+1)+x+1]
						   -unoccupyU[z*height*(width+1)+y*(width+1)+x]*du[z*height*(width+1)+y*(width+1)+x]
						   +unoccupyV[z*(height+1)*width+(y+1)*width+x]*dv[z*(height+1)*width+(y+1)*width+x]
						   +unoccupyW[(z+1)*height*width+y*width+x]*dw[(z+1)*height*width+y*width+x]
						   -unoccupyW[z*height*width+y*width+x]*dw[z*height*width+y*width+x]
						   +divergence[z*height*width+y*width+x]);
			}
			
			if(y > 0)
			{
				sigma += ratio*lambda[z*height*width+(y-1)*width+x];
				coeff += ratio2*aug_coeff;
				sigma -= ratio*aug_coeff*(
							unoccupyU[z*height*(width+1)+(y-1)*(width+1)+x+1]*du[z*height*(width+1)+(y-1)*(width+1)+x+1]
						   -unoccupyU[z*height*(width+1)+(y-1)*(width+1)+x]*du[z*height*(width+1)+(y-1)*(width+1)+x]
						   -unoccupyV[z*(height+1)*width+(y-1)*width+x]*dv[z*(height+1)*width+(y-1)*width+x]
						   +unoccupyW[(z+1)*height*width+(y-1)*width+x]*dw[(z+1)*height*width+(y-1)*width+x]
						   -unoccupyW[z*height*width+(y-1)*width+x]*dw[z*height*width+(y-1)*width+x]
						   +divergence[z*height*width+(y-1)*width+x]);
			}
			out_dv[z*(height+1)*width+y*width+x] = sigma/coeff;
		}
	}
	
	__global__
	void SolveFlux_OpenFlux_FaceRatio_w_RedBlack_Kernel(float* out_dw, const float* du, const float* dv, const float* dw, const float* unoccupyU, const float* unoccupyV, const float* unoccupyW,
										const float* divergence, const float* lambda, const float aug_coeff, const int width, const int height, const int depth, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= width || y >= height)
			return ;

		int start = redkernel ? 0 : 1;

		for(int z = start;z <= depth;z += 2)
		{
			float ratio = unoccupyW[z*height*width+y*width+x];
			
			if(ratio == 0)
				continue ;
				
			float ratio2 = ratio*ratio;
			float coeff = 2.0f*ratio,sigma = 0.0f;
			
			if(z < depth)
			{
				sigma -= ratio*lambda[z*height*width+y*width+x];
				coeff += ratio2*aug_coeff;
				sigma += ratio*aug_coeff*(
							unoccupyU[z*height*(width+1)+y*(width+1)+x+1]*du[z*height*(width+1)+y*(width+1)+x+1]
						   -unoccupyU[z*height*(width+1)+y*(width+1)+x]*du[z*height*(width+1)+y*(width+1)+x]
						   +unoccupyV[z*(height+1)*width+(y+1)*width+x]*dv[z*(height+1)*width+(y+1)*width+x]
						   -unoccupyV[z*(height+1)*width+y*width+x]*dv[z*(height+1)*width+y*width+x]
						   +unoccupyW[(z+1)*height*width+y*width+x]*dw[(z+1)*height*width+y*width+x]
						   +divergence[z*height*width+y*width+x]);
			}
			
			if(z > 0)
			{
				sigma += ratio*lambda[(z-1)*height*width+y*width+x];
				coeff += ratio2*aug_coeff;
				sigma -= ratio*aug_coeff*(
							unoccupyU[(z-1)*height*(width+1)+y*(width+1)+x+1]*du[(z-1)*height*(width+1)+y*(width+1)+x+1]
						   -unoccupyU[(z-1)*height*(width+1)+y*(width+1)+x]*du[(z-1)*height*(width+1)+y*(width+1)+x]
						   +unoccupyV[(z-1)*(height+1)*width+(y+1)*width+x]*dv[(z-1)*(height+1)*width+(y+1)*width+x]
						   -unoccupyV[(z-1)*(height+1)*width+y*width+x]*dv[(z-1)*(height+1)*width+y*width+x]						   
						   -unoccupyW[(z-1)*height*width+y*width+x]*dw[(z-1)*height*width+y*width+x]
						   +divergence[(z-1)*height*width+y*width+x]);
			}
			out_dw[z*height*width+y*width+x] = sigma/coeff;
		}
	}
										
	/********************************************************/
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	void cu_SolveOpenFluxRedBlack_MAC(float* mac_u, float* mac_v, float* mac_w, const int width, const int height, const int depth, const int outerIter, const int innerIter)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 u_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		dim3 w_gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);

		float* b_d = 0;
		float* tmp_div_d = 0;
		float* lambda_d = 0;
		checkCudaErrors( hipMalloc((void**)&b_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMalloc((void**)&lambda_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMalloc((void**)&tmp_div_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(b_d,0,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(lambda_d,0,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(tmp_div_d,0,sizeof(float)*width*height*depth));
		
		float* du_d = 0;
		float* dv_d = 0;
		float* dw_d = 0;
		float* tmp_du_d = 0;
		float* tmp_dv_d = 0;
		float* tmp_dw_d = 0;
		checkCudaErrors( hipMalloc((void**)&du_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&dv_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&dw_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMalloc((void**)&tmp_du_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&tmp_dv_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&tmp_dw_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemset(du_d,0,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMemset(dv_d,0,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMemset(dw_d,0,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemset(tmp_du_d,0,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMemset(tmp_dv_d,0,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMemset(tmp_dw_d,0,sizeof(float)*width*height*(depth+1)) );
		

		Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(b_d,mac_u,mac_v,mac_w,width,height,depth);
		
		const float max_aug_coeff = 1e6;
		float aug_coeff = 1.0f;
		for(int out_it = 0; out_it < outerIter; out_it++)
		{
			//Red-Black Solve du,dv,dw
			for(int rd_it = 0; rd_it < innerIter; rd_it++)
			{
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,dw_d,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,dw_d,b_d,lambda_d,aug_coeff,width,height,depth,false);			
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,dw_d,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,dw_d,b_d,lambda_d,aug_coeff,width,height,depth,false);
				
				checkCudaErrors( hipMemcpy(tmp_dw_d,dw_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_w_RedBlack_Kernel<<<w_gridSize,blockSize>>>(dw_d,du_d,dv_d,tmp_dw_d,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_dw_d,dw_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_w_RedBlack_Kernel<<<w_gridSize,blockSize>>>(dw_d,du_d,dv_d,tmp_dw_d,b_d,lambda_d,aug_coeff,width,height,depth,false);
			}
			
			Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(tmp_div_d,du_d,dv_d,dw_d,width,height,depth);
			
			ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<gridSize,blockSize>>>(tmp_div_d,b_d,1.0f,width,height,depth,1);
			
			ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<gridSize,blockSize>>>(lambda_d,tmp_div_d,-aug_coeff,width,height,depth,1);
			
			aug_coeff *= 2.0f;
			if(aug_coeff > max_aug_coeff)
				aug_coeff = max_aug_coeff;
		}
		
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<u_gridSize,blockSize>>>(mac_u,du_d,1.0f,width+1,height,depth,1);
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<v_gridSize,blockSize>>>(mac_v,dv_d,1.0f,width,height+1,depth,1);
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<w_gridSize,blockSize>>>(mac_w,dw_d,1.0f,width,height,depth+1,1);
		
		checkCudaErrors( hipFree(b_d) );
		checkCudaErrors( hipFree(tmp_div_d) );
		checkCudaErrors( hipFree(lambda_d) );
		checkCudaErrors( hipFree(du_d) );
		checkCudaErrors( hipFree(dv_d) );
		checkCudaErrors( hipFree(dw_d) );
		checkCudaErrors( hipFree(tmp_du_d) );
		checkCudaErrors( hipFree(tmp_dv_d) );
		checkCudaErrors( hipFree(tmp_dw_d) );
		b_d = 0;
		tmp_div_d = 0;
		lambda_d = 0;
		du_d = 0;
		dv_d = 0;
		dw_d = 0;
		tmp_du_d = 0;
		tmp_dv_d = 0;
		tmp_dw_d = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	void cu_SolveOpenFluxRedBlack_Regular(float* u, float* v, float* w, const int width, const int height, const int depth, const int outerIter, const int innerIter)
	{
		float* mac_u = 0;
		float* mac_v = 0;
		float* mac_w = 0;
		checkCudaErrors( hipMalloc((void**)&mac_u,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_v,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_w,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemset(mac_u,0,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMemset(mac_v,0,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMemset(mac_w,0,sizeof(float)*width*height*(depth+1)) );

		cu_Regular_to_MAC_vel(mac_u,mac_v,mac_w,u,v,w,width,height,depth);
		cu_SolveOpenFluxRedBlack_MAC(mac_u,mac_v,mac_w,width,height,depth,outerIter,innerIter);
		cu_MAC_to_Regular_vel(u,v,w,mac_u,mac_v,mac_w,width,height,depth);

		checkCudaErrors( hipFree(mac_u) );
		checkCudaErrors( hipFree(mac_v) );
		checkCudaErrors( hipFree(mac_w) );
		mac_u = 0;
		mac_v = 0;	
		mac_w = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	void cu_SolveOpenFluxRedBlackwithOccupy_MAC(float* mac_u, float* mac_v, float* mac_w, const bool* occupy, const int width, const int height, const int depth, const int outerIter, const int innerIter)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 u_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		dim3 w_gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);

		float* b_d = 0;
		float* tmp_div_d = 0;
		float* lambda_d = 0;
		checkCudaErrors( hipMalloc((void**)&b_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMalloc((void**)&lambda_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMalloc((void**)&tmp_div_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(b_d,0,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(lambda_d,0,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(tmp_div_d,0,sizeof(float)*width*height*depth));
		
		float* du_d = 0;
		float* dv_d = 0;
		float* dw_d = 0;
		float* tmp_du_d = 0;
		float* tmp_dv_d = 0;
		float* tmp_dw_d = 0;
		checkCudaErrors( hipMalloc((void**)&du_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&dv_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&dw_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMalloc((void**)&tmp_du_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&tmp_dv_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&tmp_dw_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemset(du_d,0,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMemset(dv_d,0,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMemset(dw_d,0,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemset(tmp_du_d,0,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMemset(tmp_dv_d,0,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMemset(tmp_dw_d,0,sizeof(float)*width*height*(depth+1)) );
		

		Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(b_d,mac_u,mac_v,mac_w,width,height,depth);
		
		float aug_coeff = 1.0f;
		float max_aug_coeff = 1e6;
		for(int out_it = 0; out_it < outerIter; out_it++)
		{
			//Red-Black Solve du,dv
			for(int rd_it = 0; rd_it < innerIter; rd_it++)
			{
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_occupy_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,dw_d,occupy,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_occupy_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,dw_d,occupy,b_d,lambda_d,aug_coeff,width,height,depth,false);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_occupy_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,dw_d,occupy,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_occupy_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,dw_d,occupy,b_d,lambda_d,aug_coeff,width,height,depth,false);
				
				checkCudaErrors( hipMemcpy(tmp_dw_d,dw_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_occupy_w_RedBlack_Kernel<<<w_gridSize,blockSize>>>(dw_d,du_d,dv_d,tmp_dw_d,occupy,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_dw_d,dw_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_occupy_w_RedBlack_Kernel<<<w_gridSize,blockSize>>>(dw_d,du_d,dv_d,tmp_dw_d,occupy,b_d,lambda_d,aug_coeff,width,height,depth,false);
			}
			
			Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(tmp_div_d,du_d,dv_d,dw_d,width,height,depth);
			
			ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<gridSize,blockSize>>>(tmp_div_d,b_d,1.0f,width,height,depth,1);
			
			ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<gridSize,blockSize>>>(lambda_d,tmp_div_d,-aug_coeff,width,height,depth,1);
			
			aug_coeff *= 2.0f;
			if(aug_coeff > max_aug_coeff)
				aug_coeff = max_aug_coeff;
		}
		
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<u_gridSize,blockSize>>>(mac_u,du_d,1.0f,width+1,height,depth,1);
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<v_gridSize,blockSize>>>(mac_v,dv_d,1.0f,width,height+1,depth,1);
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<w_gridSize,blockSize>>>(mac_w,dw_d,1.0f,width,height,depth+1,1);
		
		checkCudaErrors( hipFree(b_d) );
		checkCudaErrors( hipFree(tmp_div_d) );
		checkCudaErrors( hipFree(lambda_d) );
		checkCudaErrors( hipFree(du_d) );
		checkCudaErrors( hipFree(dv_d) );
		checkCudaErrors( hipFree(dw_d) );
		checkCudaErrors( hipFree(tmp_du_d) );
		checkCudaErrors( hipFree(tmp_dv_d) );
		checkCudaErrors( hipFree(tmp_dw_d) );
		b_d = 0;
		tmp_div_d = 0;
		lambda_d = 0;
		du_d = 0;
		dv_d = 0;
		dw_d = 0;
		tmp_du_d = 0;
		tmp_dv_d = 0;
		tmp_dw_d = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	void cu_SolveOpenFluxRedBlackwithFaceRatio_MAC(float* mac_u, float* mac_v, float* mac_w, const float* unoccupyU, const float* unoccupyV, const float* unoccupyW,
										const int width, const int height, const int depth, const int outerIter, const int innerIter)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 u_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		dim3 w_gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);


		float* b_d = 0;
		float* tmp_div_d = 0;
		float* lambda_d = 0;
		checkCudaErrors( hipMalloc((void**)&b_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMalloc((void**)&lambda_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMalloc((void**)&tmp_div_d,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(b_d,0,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(lambda_d,0,sizeof(float)*width*height*depth));
		checkCudaErrors( hipMemset(tmp_div_d,0,sizeof(float)*width*height*depth));
		
		float* du_d = 0;
		float* dv_d = 0;
		float* dw_d = 0;
		float* tmp_du_d = 0;
		float* tmp_dv_d = 0;
		float* tmp_dw_d = 0;
		checkCudaErrors( hipMalloc((void**)&du_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&dv_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&dw_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMalloc((void**)&tmp_du_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&tmp_dv_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&tmp_dw_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemset(du_d,0,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMemset(dv_d,0,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMemset(dw_d,0,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemset(tmp_du_d,0,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMemset(tmp_dv_d,0,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMemset(tmp_dw_d,0,sizeof(float)*width*height*(depth+1)) );
		

		Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(b_d,mac_u,mac_v,mac_w,width,height,depth);
		
		float aug_coeff = 1.0f;
		float max_aug_coeff = 1e6;
		for(int out_it = 0; out_it < outerIter; out_it++)
		{
			//Red-Black Solve du,dv
			for(int rd_it = 0; rd_it < innerIter; rd_it++)
			{
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_FaceRatio_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,dw_d,unoccupyU,unoccupyV,unoccupyW,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_FaceRatio_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,dw_d,unoccupyU,unoccupyV,unoccupyW,b_d,lambda_d,aug_coeff,width,height,depth,false);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_FaceRatio_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,dw_d,unoccupyU,unoccupyV,unoccupyW,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_FaceRatio_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,dw_d,unoccupyU,unoccupyV,unoccupyW,b_d,lambda_d,aug_coeff,width,height,depth,false);
				
				checkCudaErrors( hipMemcpy(tmp_dw_d,dw_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_FaceRatio_w_RedBlack_Kernel<<<w_gridSize,blockSize>>>(dw_d,du_d,dv_d,tmp_dw_d,unoccupyU,unoccupyV,unoccupyW,b_d,lambda_d,aug_coeff,width,height,depth,true);
				
				checkCudaErrors( hipMemcpy(tmp_dw_d,dw_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToDevice) );
				SolveFlux_OpenFlux_FaceRatio_w_RedBlack_Kernel<<<w_gridSize,blockSize>>>(dw_d,du_d,dv_d,tmp_dw_d,unoccupyU,unoccupyV,unoccupyW,b_d,lambda_d,aug_coeff,width,height,depth,false);
			}
			
			Calculate_Divergence_of_MAC_FaceRatio_Kernel<<<gridSize,blockSize>>>(tmp_div_d,du_d,dv_d,dw_d,unoccupyU,unoccupyV,unoccupyW,width,height,depth);
			
			ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<gridSize,blockSize>>>(tmp_div_d,b_d,1.0f,width,height,depth,1);
			
			ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<gridSize,blockSize>>>(lambda_d,tmp_div_d,-aug_coeff,width,height,depth,1);
			
			aug_coeff *= 2.0f;
			if(aug_coeff > max_aug_coeff)
				aug_coeff = max_aug_coeff;
		}
		
		
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<u_gridSize,blockSize>>>(mac_u,du_d,1.0f,width+1,height,depth,1);
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<v_gridSize,blockSize>>>(mac_v,dv_d,1.0f,width,height+1,depth,1);
		ZQ_CUDA_ImageProcessing3D::Addwith_Kernel<<<w_gridSize,blockSize>>>(mac_w,dw_d,1.0f,width,height,depth+1,1);
		
		checkCudaErrors( hipFree(b_d) );
		checkCudaErrors( hipFree(tmp_div_d) );
		checkCudaErrors( hipFree(lambda_d) );
		checkCudaErrors( hipFree(du_d) );
		checkCudaErrors( hipFree(dv_d) );
		checkCudaErrors( hipFree(dw_d) );
		checkCudaErrors( hipFree(tmp_du_d) );
		checkCudaErrors( hipFree(tmp_dv_d) );
		checkCudaErrors( hipFree(tmp_dw_d) );
		b_d = 0;
		tmp_div_d = 0;
		lambda_d = 0;
		du_d = 0;
		dv_d = 0;
		dw_d = 0;
		tmp_du_d = 0;
		tmp_dv_d = 0;
		tmp_dw_d = 0;
	}
	
	
	/*************************************************************/
	
	/*First Implementation*/
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	extern "C" 
	void SolveOpenFluxRedBlack3D_MAC(float* mac_u, float* mac_v, float* mac_w, const int width, const int height, const int depth, const int outerIter, const int innerIter)
	{
		float* mac_u_d = 0;
		float* mac_v_d = 0;
		float* mac_w_d = 0;

		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_w_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1)*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_w_d,mac_w,sizeof(float)*width*height*(depth+1),hipMemcpyHostToDevice) );

		cu_SolveOpenFluxRedBlack_MAC(mac_u_d,mac_v_d,mac_w_d,width,height,depth,outerIter,innerIter);

		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_w,mac_w_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		checkCudaErrors( hipFree(mac_w_d) );
		mac_u_d = 0;
		mac_v_d = 0;
		mac_w_d = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	extern "C"
	void SolveOpenFluxRedBlack3D_Regular(float* u, float* v, float* w, const int width, const int height, const int depth, const int outerIter, const int innerIter)
	{
		float* u_d = 0;
		float* v_d = 0;
		float* w_d = 0;

		checkCudaErrors( hipMalloc((void**)&u_d,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&v_d,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&w_d,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemcpy(u_d,u,sizeof(float)*width*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(v_d,v,sizeof(float)*width*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(w_d,w,sizeof(float)*width*height*depth,hipMemcpyHostToDevice) );

		cu_SolveOpenFluxRedBlack_Regular(u_d,v_d,w_d,width,height,depth,outerIter,innerIter);

		checkCudaErrors( hipMemcpy(u,u_d,sizeof(float)*width*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(v,v_d,sizeof(float)*width*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(w,w_d,sizeof(float)*width*height*depth,hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(u_d) );
		checkCudaErrors( hipFree(v_d) );
		checkCudaErrors( hipFree(w_d) );
		u_d = 0;
		v_d = 0;
		w_d = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	extern "C" 
	void SolveOpenFluxRedBlackwithOccupy3D_MAC(float* mac_u, float* mac_v, float* mac_w, const bool* occupy, const int width, const int height, const int depth,
									const int outerIter, const int innerIter)
	{
		float* mac_u_d = 0;
		float* mac_v_d = 0;
		float* mac_w_d = 0;
		bool* occupy_d = 0;

		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_w_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMalloc((void**)&occupy_d,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1)*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_w_d,mac_w,sizeof(float)*width*height*(depth+1),hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(occupy_d,occupy,sizeof(bool)*width*height*depth,hipMemcpyHostToDevice) );

		cu_SolveOpenFluxRedBlackwithOccupy_MAC(mac_u_d,mac_v_d,mac_w_d,occupy_d,width,height,depth,outerIter,innerIter);

		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_w,mac_w_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		checkCudaErrors( hipFree(mac_w_d) );
		checkCudaErrors( hipFree(occupy_d) );
		mac_u_d = 0;
		mac_v_d = 0;
		mac_w_d = 0;
		occupy_d = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	extern "C" 
	void SolveOpenFluxRedBlackwithFaceRatio3D_MAC(float* mac_u, float* mac_v, float* mac_w, const float* unoccupyU, const float* unoccupyV, const float* unoccupyW,
											const int width, const int height, const int depth, const int outerIter, const int innerIter)
	{
		float* mac_u_d = 0;
		float* mac_v_d = 0;
		float* mac_w_d = 0;
		float* unoccupyU_d = 0;
		float* unoccupyV_d = 0;
		float* unoccupyW_d = 0;

		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_w_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMalloc((void**)&unoccupyU_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&unoccupyV_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&unoccupyW_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1)*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_w_d,mac_w,sizeof(float)*width*height*(depth+1),hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(unoccupyU_d,unoccupyU,sizeof(float)*(width+1)*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(unoccupyV_d,unoccupyV,sizeof(float)*width*(height+1)*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(unoccupyW_d,unoccupyW,sizeof(float)*width*height*(depth+1),hipMemcpyHostToDevice) );
		
		cu_SolveOpenFluxRedBlackwithFaceRatio_MAC(mac_u_d,mac_v_d,mac_w_d,unoccupyU_d,unoccupyV_d,unoccupyW_d,width,height,depth,outerIter,innerIter);

		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_w,mac_w_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		checkCudaErrors( hipFree(mac_w_d) );
		checkCudaErrors( hipFree(unoccupyU_d) );
		checkCudaErrors( hipFree(unoccupyV_d) );
		checkCudaErrors( hipFree(unoccupyW_d) );
		mac_u_d = 0;
		mac_v_d = 0;
		mac_w_d = 0;
		unoccupyU_d = 0;
		unoccupyV_d = 0;
		unoccupyW_d = 0;
	}
	
	
}

#endif