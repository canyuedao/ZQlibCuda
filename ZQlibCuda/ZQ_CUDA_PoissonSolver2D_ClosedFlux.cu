#include "hip/hip_runtime.h"
#ifndef _ZQ_CUDA_POISSON_SOLVER_2D_CLOSED_FLUX_CU_
#define _ZQ_CUDA_POISSON_SOLVER_2D_CLOSED_FLUX_CU_

#include "ZQ_CUDA_PoissonSolver2D_ClosedFlux.cuh"
#include "ZQ_CUDA_ImageProcessing2D.cuh"

namespace ZQ_CUDA_PoissonSolver2D
{
	__global__
	void SolveFlux_ClosedFlux_u_RedBlack_Kernel(float* out_du, const float* du, const float* dv, const float* divergence, const float* lambda, const float aug_coeff,
										const float div_per_volume, const int width, const int height, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x; // x is in [0, width-2]
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		
		x = x + 1; // then x is in [1, width-1]

		if(x >= width || y >= height)
			return ;

		int rest = x%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		float coeff = 2.0f,sigma = 0.0f;
		
		sigma -= lambda[y*width+x];
		coeff += aug_coeff;
		sigma += aug_coeff*(du[y*(width+1)+x+1]+dv[(y+1)*width+x]-dv[y*width+x]+divergence[y*width+x]-div_per_volume);
		
		sigma += lambda[y*width+x-1];
		coeff += aug_coeff;
		sigma -= aug_coeff*(-du[y*(width+1)+x-1]+dv[(y+1)*width+x-1]-dv[y*width+x-1]+divergence[y*width+x-1]-div_per_volume);
		
		out_du[y*(width+1)+x] = sigma/coeff;
	}
	
	
	__global__
	void SolveFlux_ClosedFlux_v_RedBlack_Kernel(float* out_dv, const float* du, const float* dv, const float* divergence, const float* lambda, const float aug_coeff,
										const float div_per_volume, const int width, const int height, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y; //y is in [0,height-2]

		y = y + 1;	// y is in [1, height-1]
		
		if(x >= width || y >= height)
			return ;

		int rest = y%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		float coeff = 2.0f,sigma = 0.0f;
		sigma -= lambda[y*width+x];
		coeff += aug_coeff;
		sigma += aug_coeff*(du[y*(width+1)+x+1]-du[y*(width+1)+x]+dv[(y+1)*width+x]+divergence[y*width+x]-div_per_volume);
		sigma += lambda[(y-1)*width+x];
		coeff += aug_coeff;
		sigma -= aug_coeff*(du[(y-1)*(width+1)+x+1]-du[(y-1)*(width+1)+x]-dv[(y-1)*width+x]+divergence[(y-1)*width+x] - div_per_volume);
		
		out_dv[y*width+x] = sigma/coeff;
	}
	
	__global__
	void SolveFlux_ClosedFlux_occupy_u_RedBlack_Kernel(float* out_du, const float* du, const float* dv, const bool* occupy, const float* divergence, const float* lambda, const float aug_coeff,
										const float div_per_volume, const int width, const int height, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x; //x is in[0, width-2]
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		x = x + 1; // then x s in [1,width-1]
		if(x >= width || y >= height)
			return ;

		int rest = x%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		float coeff = 2.0f,sigma = 0.0f;
		
		if(occupy[y*width+x])
			return ;
		sigma -= lambda[y*width+x];
		coeff += aug_coeff;
		sigma += aug_coeff*(du[y*(width+1)+x+1]+dv[(y+1)*width+x]-dv[y*width+x]+divergence[y*width+x] - div_per_volume);
		
		if(occupy[y*width+x-1])
			return ;
		sigma += lambda[y*width+x-1];
		coeff += aug_coeff;
		sigma -= aug_coeff*(-du[y*(width+1)+x-1]+dv[(y+1)*width+x-1]-dv[y*width+x-1]+divergence[y*width+x-1] - div_per_volume);
		
		out_du[y*(width+1)+x] = sigma/coeff;
	}
	
	
	__global__
	void SolveFlux_ClosedFlux_occupy_v_RedBlack_Kernel(float* out_dv, const float* du, const float* dv, const bool* occupy, const float* divergence, const float* lambda, const float aug_coeff,
										const float div_per_volume, const int width, const int height, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y; // y is in [0, height-2]

		y = y + 1; // y is in [1,height-1]
		if(x >= width || y >= height)
			return ;

		int rest = y%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		float coeff = 2.0f,sigma = 0.0f;
		
		if(occupy[y*width+x])
			return ;
		sigma -= lambda[y*width+x];
		coeff += aug_coeff;
		sigma += aug_coeff*(du[y*(width+1)+x+1]-du[y*(width+1)+x]+dv[(y+1)*width+x]+divergence[y*width+x]-div_per_volume);
		
		if(occupy[(y-1)*width+x])
			return ;
		sigma += lambda[(y-1)*width+x];
		coeff += aug_coeff;
		sigma -= aug_coeff*(du[(y-1)*(width+1)+x+1]-du[(y-1)*(width+1)+x]-dv[(y-1)*width+x]+divergence[(y-1)*width+x]-div_per_volume);
		
		out_dv[y*width+x] = sigma/coeff;
	}
	
	__global__
	void SolveFlux_ClosedFlux_FaceRatio_u_RedBlack_Kernel(float* out_du, const float* du, const float* dv, const float* unoccupyVolume, const float* unoccupyU, const float* unoccupyV,
										const float* divergence, const float* lambda, const float aug_coeff, 
										const float div_per_volume, const int width, const int height, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x; // x is in [0,width-2]
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		x = x + 1; // x is in [1,width-1]
		if(x >= width || y >= height)
			return ;

		int rest = x%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		float ratio = unoccupyU[y*(width+1)+x];
		float ratio2 = ratio*ratio;
		
		if(ratio == 0)
			return ;
		
		float coeff = 2.0f*ratio,sigma = 0.0f;
		
		sigma -= ratio*lambda[y*width+x];
		coeff += ratio2*aug_coeff;
		sigma += ratio*aug_coeff*(
					unoccupyU[y*(width+1)+x+1]*du[y*(width+1)+x+1]
				   +unoccupyV[(y+1)*width+x]*dv[(y+1)*width+x]
				   -unoccupyV[y*width+x]*dv[y*width+x]
				   +divergence[y*width+x]-div_per_volume*unoccupyVolume[y*width+x]);
		
		sigma += ratio*lambda[y*width+x-1];
		coeff += ratio2*aug_coeff;
		sigma -= ratio*aug_coeff*(
					-unoccupyU[y*(width+1)+x-1]*du[y*(width+1)+x-1]
					+unoccupyV[(y+1)*width+x-1]*dv[(y+1)*width+x-1]
					-unoccupyV[y*width+x-1]*dv[y*width+x-1]
					+divergence[y*width+x-1]-div_per_volume*unoccupyVolume[y*width+x-1]);
		out_du[y*(width+1)+x] = sigma/coeff;
	}
	
	__global__
	void SolveFlux_ClosedFlux_FaceRatio_v_RedBlack_Kernel(float* out_dv, const float* du, const float* dv, const float* unoccupyVolume, const float* unoccupyU, const float* unoccupyV,
										const float* divergence, const float* lambda, const float aug_coeff, 
										const float div_per_volume, const int width, const int height, const bool redkernel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y; // y is in [0,height-2]

		y = y + 1; //then y is in [1,height-1]
		if(x >= width || y >= height)
			return ;

		int rest = y%2;

		if(rest == (redkernel ? 1 : 0))
			return;

		float ratio = unoccupyV[y*width+x];
		
		if(ratio == 0)
			return ;
			
		float ratio2 = ratio*ratio;
		float coeff = 2.0f*ratio,sigma = 0.0f;
		
		sigma -= ratio*lambda[y*width+x];
		coeff += ratio2*aug_coeff;
		sigma += ratio*aug_coeff*(
					unoccupyU[y*(width+1)+x+1]*du[y*(width+1)+x+1]
				   -unoccupyU[y*(width+1)+x]*du[y*(width+1)+x]
				   +unoccupyV[(y+1)*width+x]*dv[(y+1)*width+x]
				   +divergence[y*width+x]-div_per_volume*unoccupyVolume[y*width+x]);
		
		sigma += ratio*lambda[(y-1)*width+x];
		coeff += ratio2*aug_coeff;
		sigma -= ratio*aug_coeff*(
					unoccupyU[(y-1)*(width+1)+x+1]*du[(y-1)*(width+1)+x+1]
				   -unoccupyU[(y-1)*(width+1)+x]*du[(y-1)*(width+1)+x]
				   -unoccupyV[(y-1)*width+x]*dv[(y-1)*width+x]
				   +divergence[(y-1)*width+x]-div_per_volume*unoccupyVolume[(y-1)*width+x]);
		
		out_dv[y*width+x] = sigma/coeff;
	}
	
	/********************************************************/
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	void cu_SolveClosedFluxRedBlack_MAC(float* mac_u, float* mac_v, const float div_per_volume, const int width, const int height, const int outerIter, const int innerIter)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 u_gridSize((width-1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height-1+blockSize.y-1)/blockSize.y);


		float* b_d = 0;
		float* tmp_div_d = 0;
		float* lambda_d = 0;
		checkCudaErrors( hipMalloc((void**)&b_d,sizeof(float)*width*height));
		checkCudaErrors( hipMalloc((void**)&lambda_d,sizeof(float)*width*height));
		checkCudaErrors( hipMalloc((void**)&tmp_div_d,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(b_d,0,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(lambda_d,0,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(tmp_div_d,0,sizeof(float)*width*height));
		
		float* du_d = 0;
		float* dv_d = 0;
		float* tmp_du_d = 0;
		float* tmp_dv_d = 0;
		checkCudaErrors( hipMalloc((void**)&du_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&dv_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMalloc((void**)&tmp_du_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&tmp_dv_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemset(du_d,0,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMemset(dv_d,0,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemset(tmp_du_d,0,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMemset(tmp_dv_d,0,sizeof(float)*width*(height+1)) );
		

		Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(b_d,mac_u,mac_v,width,height);
		
		float aug_coeff = 1.0f;
		for(int out_it = 0; out_it < outerIter; out_it++)
		{
			//Red-Black Solve du,dv
			for(int rd_it = 0; rd_it < innerIter; rd_it++)
			{
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,b_d,lambda_d,aug_coeff,div_per_volume,width,height,true);
				
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,b_d,lambda_d,aug_coeff,div_per_volume,width,height,false);			
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,b_d,lambda_d,aug_coeff,div_per_volume,width,height,true);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,b_d,lambda_d,aug_coeff,div_per_volume,width,height,false);
			}
			
			Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(tmp_div_d,du_d,dv_d,width,height);
			
			ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<gridSize,blockSize>>>(tmp_div_d,b_d,1.0f,width,height,1);
			
			ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<gridSize,blockSize>>>(lambda_d,tmp_div_d,-aug_coeff,width,height,1);
			
			aug_coeff *= 2.0f;
		}
		
		dim3 uu_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 vv_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<uu_gridSize,blockSize>>>(mac_u,du_d,1.0f,width+1,height,1);
		ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<vv_gridSize,blockSize>>>(mac_v,dv_d,1.0f,width,height+1,1);
		
		checkCudaErrors( hipFree(b_d) );
		checkCudaErrors( hipFree(tmp_div_d) );
		checkCudaErrors( hipFree(lambda_d) );
		checkCudaErrors( hipFree(du_d) );
		checkCudaErrors( hipFree(dv_d) );
		checkCudaErrors( hipFree(tmp_du_d) );
		checkCudaErrors( hipFree(tmp_dv_d) );
		b_d = 0;
		tmp_div_d = 0;
		lambda_d = 0;
		du_d = 0;
		dv_d = 0;
		tmp_du_d = 0;
		tmp_dv_d = 0;
	}
	
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	void cu_SolveClosedFluxRedBlackwithOccupy_MAC(float* mac_u, float* mac_v, const bool* occupy, const float div_per_volume, const int width, const int height, const int outerIter, const int innerIter)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 u_gridSize((width-1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height-1+blockSize.y-1)/blockSize.y);


		float* b_d = 0;
		float* tmp_div_d = 0;
		float* lambda_d = 0;
		checkCudaErrors( hipMalloc((void**)&b_d,sizeof(float)*width*height));
		checkCudaErrors( hipMalloc((void**)&lambda_d,sizeof(float)*width*height));
		checkCudaErrors( hipMalloc((void**)&tmp_div_d,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(b_d,0,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(lambda_d,0,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(tmp_div_d,0,sizeof(float)*width*height));
		
		float* du_d = 0;
		float* dv_d = 0;
		float* tmp_du_d = 0;
		float* tmp_dv_d = 0;
		checkCudaErrors( hipMalloc((void**)&du_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&dv_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMalloc((void**)&tmp_du_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&tmp_dv_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemset(du_d,0,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMemset(dv_d,0,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemset(tmp_du_d,0,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMemset(tmp_dv_d,0,sizeof(float)*width*(height+1)) );
		

		Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(b_d,mac_u,mac_v,width,height);
		
		float aug_coeff = 1.0f;
		float max_aug_coeff = 1e6;
		for(int out_it = 0; out_it < outerIter; out_it++)
		{
			//Red-Black Solve du,dv
			for(int rd_it = 0; rd_it < innerIter; rd_it++)
			{
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_occupy_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,occupy,b_d,lambda_d,aug_coeff,div_per_volume,width,height,true);
				
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_occupy_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,occupy,b_d,lambda_d,aug_coeff,div_per_volume,width,height,false);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_occupy_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,occupy,b_d,lambda_d,aug_coeff,div_per_volume,width,height,true);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_occupy_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,occupy,b_d,lambda_d,aug_coeff,div_per_volume,width,height,false);
			}
			
			Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(tmp_div_d,du_d,dv_d,width,height);
			
			ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<gridSize,blockSize>>>(tmp_div_d,b_d,1.0f,width,height,1);
			
			ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<gridSize,blockSize>>>(lambda_d,tmp_div_d,-aug_coeff,width,height,1);
			
			aug_coeff *= 2.0f;
			if(aug_coeff > max_aug_coeff)
				aug_coeff = max_aug_coeff;
		}
		
		dim3 uu_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 vv_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<uu_gridSize,blockSize>>>(mac_u,du_d,1.0f,width+1,height,1);
		ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<vv_gridSize,blockSize>>>(mac_v,dv_d,1.0f,width,height+1,1);
		
		checkCudaErrors( hipFree(b_d) );
		checkCudaErrors( hipFree(tmp_div_d) );
		checkCudaErrors( hipFree(lambda_d) );
		checkCudaErrors( hipFree(du_d) );
		checkCudaErrors( hipFree(dv_d) );
		checkCudaErrors( hipFree(tmp_du_d) );
		checkCudaErrors( hipFree(tmp_dv_d) );
		b_d = 0;
		tmp_div_d = 0;
		lambda_d = 0;
		du_d = 0;
		dv_d = 0;
		tmp_du_d = 0;
		tmp_dv_d = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	void cu_SolveClosedFluxRedBlackwithFaceRatio_MAC(float* mac_u, float* mac_v, const float* unoccupyVolume, const float* unoccupyU, const float* unoccupyV,
										const float div_per_volume, const int width, const int height, const int outerIter, const int innerIter)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 u_gridSize((width-1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height-1+blockSize.y-1)/blockSize.y);


		float* b_d = 0;
		float* tmp_div_d = 0;
		float* lambda_d = 0;
		checkCudaErrors( hipMalloc((void**)&b_d,sizeof(float)*width*height));
		checkCudaErrors( hipMalloc((void**)&lambda_d,sizeof(float)*width*height));
		checkCudaErrors( hipMalloc((void**)&tmp_div_d,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(b_d,0,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(lambda_d,0,sizeof(float)*width*height));
		checkCudaErrors( hipMemset(tmp_div_d,0,sizeof(float)*width*height));
		
		float* du_d = 0;
		float* dv_d = 0;
		float* tmp_du_d = 0;
		float* tmp_dv_d = 0;
		checkCudaErrors( hipMalloc((void**)&du_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&dv_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMalloc((void**)&tmp_du_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&tmp_dv_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemset(du_d,0,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMemset(dv_d,0,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemset(tmp_du_d,0,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMemset(tmp_dv_d,0,sizeof(float)*width*(height+1)) );
		

		Calculate_Divergence_of_MAC_Kernel<<<gridSize,blockSize>>>(b_d,mac_u,mac_v,width,height);
		
		float aug_coeff = 1.0f;
		float max_aug_coeff = 1e6;
		for(int out_it = 0; out_it < outerIter; out_it++)
		{
			//Red-Black Solve du,dv
			for(int rd_it = 0; rd_it < innerIter; rd_it++)
			{
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_FaceRatio_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,unoccupyVolume,unoccupyU,unoccupyV,b_d,lambda_d,aug_coeff,
																							div_per_volume,width,height,true);
				
				checkCudaErrors( hipMemcpy(tmp_du_d,du_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_FaceRatio_u_RedBlack_Kernel<<<u_gridSize,blockSize>>>(du_d,tmp_du_d,dv_d,unoccupyVolume,unoccupyU,unoccupyV,b_d,lambda_d,aug_coeff,
																							div_per_volume,width,height,false);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_FaceRatio_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,unoccupyVolume,unoccupyU,unoccupyV,b_d,lambda_d,aug_coeff,
																							div_per_volume,width,height,true);
				
				checkCudaErrors( hipMemcpy(tmp_dv_d,dv_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToDevice) );
				SolveFlux_ClosedFlux_FaceRatio_v_RedBlack_Kernel<<<v_gridSize,blockSize>>>(dv_d,du_d,tmp_dv_d,unoccupyVolume,unoccupyU,unoccupyV,b_d,lambda_d,aug_coeff,
																							div_per_volume,width,height,false);
			}
			
			Calculate_Divergence_of_MAC_FaceRatio_Kernel<<<gridSize,blockSize>>>(tmp_div_d,du_d,dv_d,unoccupyU,unoccupyV,width,height);
			
			ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<gridSize,blockSize>>>(tmp_div_d,b_d,1.0f,width,height,1);
			
			ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<gridSize,blockSize>>>(tmp_div_d,unoccupyVolume,-div_per_volume,width,height,1);
			
			ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<gridSize,blockSize>>>(lambda_d,tmp_div_d,-aug_coeff,width,height,1);
			
			aug_coeff *= 2.0f;
			if(aug_coeff > max_aug_coeff)
				aug_coeff = max_aug_coeff;
		}
		
		dim3 uu_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 vv_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<uu_gridSize,blockSize>>>(mac_u,du_d,1.0f,width+1,height,1);
		ZQ_CUDA_ImageProcessing2D::Addwith_Kernel<<<vv_gridSize,blockSize>>>(mac_v,dv_d,1.0f,width,height+1,1);
		
		checkCudaErrors( hipFree(b_d) );
		checkCudaErrors( hipFree(tmp_div_d) );
		checkCudaErrors( hipFree(lambda_d) );
		checkCudaErrors( hipFree(du_d) );
		checkCudaErrors( hipFree(dv_d) );
		checkCudaErrors( hipFree(tmp_du_d) );
		checkCudaErrors( hipFree(tmp_dv_d) );
		b_d = 0;
		tmp_div_d = 0;
		lambda_d = 0;
		du_d = 0;
		dv_d = 0;
		tmp_du_d = 0;
		tmp_dv_d = 0;
	}
	
	/*************************************************************/
	
	/*First Implementation*/
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	extern "C" 
	void SolveClosedFluxRedBlack2D_MAC(float* mac_u, float* mac_v, const float div_per_volume, const int width, const int height, const int outerIter, const int innerIter)
	{
		float* mac_u_d = 0;
		float* mac_v_d = 0;

		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1),hipMemcpyHostToDevice) );

		cu_SolveClosedFluxRedBlack_MAC(mac_u_d,mac_v_d,div_per_volume,width,height,outerIter,innerIter);

		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		mac_u_d = 0;
		mac_v_d = 0;
	}
	
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	extern "C" 
	void SolveClosedFluxRedBlackwithOccupy2D_MAC(float* mac_u, float* mac_v, const bool* occupy, const float div_per_volume, const int width, const int height, const int outerIter, const int innerIter)
	{
		float* mac_u_d = 0;
		float* mac_v_d = 0;
		bool* occupy_d = 0;

		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMalloc((void**)&occupy_d,sizeof(float)*width*height) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1),hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(occupy_d,occupy,sizeof(bool)*width*height,hipMemcpyHostToDevice) );

		cu_SolveClosedFluxRedBlackwithOccupy_MAC(mac_u_d,mac_v_d,occupy_d,div_per_volume,width,height,outerIter,innerIter);

		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		checkCudaErrors( hipFree(occupy_d) );
		mac_u_d = 0;
		mac_v_d = 0;
		occupy_d = 0;
	}
	
	/*outer iteration: Augmented Lagrange Multiplier method
	* inner iteration: red-black iteration
	*/
	extern "C" 
	void SolveClosedFluxRedBlackwithFaceRatio2D_MAC(float* mac_u, float* mac_v, const float* unoccupyVolume, const float* unoccupyU, const float* unoccupyV,
											const float div_per_volume, const int width, const int height, const int outerIter, const int innerIter)
	{
		float* mac_u_d = 0;
		float* mac_v_d = 0;
		float* unoccupyVolume_d = 0;
		float* unoccupyU_d = 0;
		float* unoccupyV_d = 0;

		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMalloc((void**)&unoccupyVolume_d,sizeof(float)*width*height) );
		checkCudaErrors( hipMalloc((void**)&unoccupyU_d,sizeof(float)*(width+1)*height) );
		checkCudaErrors( hipMalloc((void**)&unoccupyV_d,sizeof(float)*width*(height+1)) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1),hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(unoccupyVolume_d,unoccupyVolume,sizeof(float)*width*height,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(unoccupyU_d,unoccupyU,sizeof(float)*(width+1)*height,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(unoccupyV_d,unoccupyV,sizeof(float)*width*(height+1),hipMemcpyHostToDevice) );
		
		cu_SolveClosedFluxRedBlackwithFaceRatio_MAC(mac_u_d,mac_v_d,unoccupyVolume_d,unoccupyU_d,unoccupyV_d,div_per_volume,width,height,outerIter,innerIter);

		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1),hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		checkCudaErrors( hipFree(unoccupyVolume_d) );
		checkCudaErrors( hipFree(unoccupyU_d) );
		checkCudaErrors( hipFree(unoccupyV_d) );
		mac_u_d = 0;
		mac_v_d = 0;
		unoccupyVolume_d = 0;
		unoccupyU_d = 0;
		unoccupyV_d = 0;
	}
}

#endif